#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

#include "reduce.cuh"

__global__ void k_harm(int N, float freq, float* acc) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float e = 0.0f;
    if (i < N) {
        float t = (float)i / (float)N;
        float y = __sinf(2.0f * HIP_PI_F * freq * t);
        e = y * y;
    }
    float s = block_sum<256>(e);
    if (threadIdx.x == 0) {
        atomicAdd(acc, s);
    }
}

extern "C" int qallow_p13_harmonic_gpu(int N, float freq, float* out_energy) {
    float* d_acc = nullptr;
    hipError_t err = hipMalloc(&d_acc, sizeof(float));
    if (err != hipSuccess) {
        return -1;
    }
    hipMemset(d_acc, 0, sizeof(float));

    dim3 bs(256);
    dim3 gs((N + bs.x - 1) / bs.x);
    k_harm<<<gs, bs>>>(N, freq, d_acc);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        hipFree(d_acc);
        return -2;
    }

    float h = 0.0f;
    hipMemcpy(&h, d_acc, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_acc);
    *out_energy = h;
    return 0;
}
