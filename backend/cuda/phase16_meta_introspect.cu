#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>

#include "core/phase16_meta_introspect.cuh"

extern "C" __global__
void introspect_kernel(const float* __restrict__ dur,
                       const float* __restrict__ coh,
                       const float* __restrict__ eth,
                       float* __restrict__ out,
                       int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float duration = fmaxf(dur[i], 0.0f);
        float coherence = coh[i];
        float ethics = eth[i];
        float score = 0.4f * coherence + 0.4f * ethics + 0.2f * log1pf(duration);
        out[i] = fminf(score, 1.0f);
    }
}

extern "C" int qallow_meta_introspect_gpu(const float* durations,
                                          const float* coherence,
                                          const float* ethics,
                                          float* improvement_scores,
                                          int count) {
    if (!durations || !coherence || !ethics || !improvement_scores || count <= 0) {
        return -1;
    }

    float *d_durations = nullptr, *d_coherence = nullptr, *d_ethics = nullptr, *d_out = nullptr;
    size_t bytes = sizeof(float) * (size_t)count;

    if (hipMalloc(&d_durations, bytes) != hipSuccess ||
        hipMalloc(&d_coherence, bytes) != hipSuccess ||
        hipMalloc(&d_ethics, bytes) != hipSuccess ||
        hipMalloc(&d_out, bytes) != hipSuccess) {
        hipFree(d_durations);
        hipFree(d_coherence);
        hipFree(d_ethics);
        hipFree(d_out);
        return -2;
    }

    hipMemcpy(d_durations, durations, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_coherence, coherence, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_ethics, ethics, bytes, hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((count + block.x - 1) / block.x);
    introspect_kernel<<<grid, block>>>(d_durations, d_coherence, d_ethics, d_out, count);
    hipError_t sync_err = hipDeviceSynchronize();
    if (sync_err != hipSuccess) {
        hipFree(d_durations);
        hipFree(d_coherence);
        hipFree(d_ethics);
        hipFree(d_out);
        return -3;
    }

    hipMemcpy(improvement_scores, d_out, bytes, hipMemcpyDeviceToHost);

    hipFree(d_durations);
    hipFree(d_coherence);
    hipFree(d_ethics);
    hipFree(d_out);
    return 0;
}
