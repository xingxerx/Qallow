#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include "pocket.h"

#define CUDA_OK(x) do{auto e=(x); if(e!=hipSuccess){ \
  fprintf(stderr,"CUDA error %s:%d: %s\n",__FILE__,__LINE__,hipGetErrorString(e)); return -1;}}while(0)

static int    G_P=0, G_N=0;
static double *d_orb=nullptr, *d_riv=nullptr, *d_myc=nullptr;              // [P*N]
static double *d_orb_mean=nullptr, *d_riv_mean=nullptr, *d_myc_mean=nullptr; // [N]

// simple per-thread RNG: LCG
__device__ inline double lcg(uint32_t &s){
  s = 1664525u*s + 1013904223u;
  return (double)(s & 0x00FFFFFF) / (double)0x01000000; // [0,1)
}

// init pocket states
__global__ void k_init(double* O, double* R, double* M, int P, int N){
  int i = blockIdx.x*blockDim.x + threadIdx.x;   // node
  int p = blockIdx.y*blockDim.y + threadIdx.y;   // pocket
  if(p>=P||i>=N) return;
  int idx = p*N + i;
  O[idx] = 0.9342;  // seed near your observed bands
  R[idx] = 0.9991;
  M[idx] = 1.0000;
}

// one tick update
__global__ void k_update(double* O, double* R, double* M, int P, int N, double jitter, int t){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int p = blockIdx.y*blockDim.y + threadIdx.y;
  if(p>=P||i>=N) return;
  int idx = p*N + i;
  // cheap stochastic dynamics
  uint32_t seed = (1234567u ^ (p*73856093u) ^ (i*19349663u) ^ (t*83492791u));
  double j = (lcg(seed)-0.5)*2.0*jitter;   // [-jitter, +jitter]
  double o = O[idx] + j*0.8;
  double r = R[idx] - 0.0002 + j*0.1;
  double m = M[idx] - 0.00001 + j*0.05;
  // clamp
  o = fmin(fmax(o, 0.90), 0.95);
  r = fmin(fmax(r, 0.995),1.000);
  m = fmin(fmax(m, 0.9995),1.0000);
  O[idx]=o; R[idx]=r; M[idx]=m;
}

// reduction: mean over pockets for each node
__global__ void k_mean_over_pockets(const double* __restrict__ X, double* __restrict__ OUT, int P, int N){
  int i = blockIdx.x*blockDim.x + threadIdx.x; // node
  if(i>=N) return;
  double acc=0.0;
  for(int p=0;p<P;++p) acc += X[p*N + i];
  OUT[i] = acc / (double)P;
}

int pocket_spawn_and_run(const pocket_cfg_t* cfg){
  if(!cfg || cfg->pockets<=0 || cfg->nodes<=0 || cfg->steps<=0) return -1;
  G_P = cfg->pockets; G_N = cfg->nodes;

  size_t PN = (size_t)G_P * (size_t)G_N;
  CUDA_OK(hipMalloc(&d_orb, PN*sizeof(double)));
  CUDA_OK(hipMalloc(&d_riv, PN*sizeof(double)));
  CUDA_OK(hipMalloc(&d_myc, PN*sizeof(double)));
  CUDA_OK(hipMalloc(&d_orb_mean, G_N*sizeof(double)));
  CUDA_OK(hipMalloc(&d_riv_mean, G_N*sizeof(double)));
  CUDA_OK(hipMalloc(&d_myc_mean, G_N*sizeof(double)));

  dim3 block(128,1);              // nodes x pockets split
  dim3 grid((G_N+block.x-1)/block.x, G_P); // one warp-row per pocket

  k_init<<<grid, block>>>(d_orb, d_riv, d_myc, G_P, G_N);
  CUDA_OK(hipGetLastError());

  // stream per pocket (optional; kernel already 2D). Example multi-stream loop:
  // Here we keep single kernel per tick for simplicity and good occupancy.
  for(int t=0;t<cfg->steps;++t){
    k_update<<<grid, block>>>(d_orb, d_riv, d_myc, G_P, G_N, cfg->jitter, t);
    CUDA_OK(hipGetLastError());
  }
  CUDA_OK(hipDeviceSynchronize());

  // means per node
  dim3 block1(256);
  dim3 grid1((G_N+block1.x-1)/block1.x);
  k_mean_over_pockets<<<grid1, block1>>>(d_orb, d_orb_mean, G_P, G_N);
  k_mean_over_pockets<<<grid1, block1>>>(d_riv, d_riv_mean, G_P, G_N);
  k_mean_over_pockets<<<grid1, block1>>>(d_myc, d_myc_mean, G_P, G_N);
  CUDA_OK(hipGetLastError());
  CUDA_OK(hipDeviceSynchronize());
  return 0;
}

int pocket_merge_to_host(double* orbital, double* river, double* mycelial){
  if(!d_orb_mean||!d_riv_mean||!d_myc_mean) return -1;
  CUDA_OK(hipMemcpy(orbital,  d_orb_mean, G_N*sizeof(double), hipMemcpyDeviceToHost));
  CUDA_OK(hipMemcpy(river,    d_riv_mean, G_N*sizeof(double), hipMemcpyDeviceToHost));
  CUDA_OK(hipMemcpy(mycelial, d_myc_mean, G_N*sizeof(double), hipMemcpyDeviceToHost));
  return 0;
}

int pocket_release(){
  if(d_orb)       hipFree(d_orb), d_orb=nullptr;
  if(d_riv)       hipFree(d_riv), d_riv=nullptr;
  if(d_myc)       hipFree(d_myc), d_myc=nullptr;
  if(d_orb_mean)  hipFree(d_orb_mean), d_orb_mean=nullptr;
  if(d_riv_mean)  hipFree(d_riv_mean), d_riv_mean=nullptr;
  if(d_myc_mean)  hipFree(d_myc_mean), d_myc_mean=nullptr;
  G_P=G_N=0;
  return 0;
}
