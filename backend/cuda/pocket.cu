#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdint>
#include <stdio.h>
extern "C" {
#include "qallow.h"
#include "pocket.h"
}

// CUDA error checking macro
#define CUDA_OK(call) do { \
    hipError_t e = (call); \
    if(e != hipSuccess) { \
        fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        return -1; \
    } \
} while(0)

// Global state
static double *d_orb = nullptr, *d_riv = nullptr, *d_myc = nullptr;
static double *d_orb_mean = nullptr, *d_riv_mean = nullptr, *d_myc_mean = nullptr;
static int G_P = 0, G_N = 0;

/* simple linear congruential generator */
static __device__ __inline__ double lcg(uint32_t* s){
    *s = (*s * 1664525u + 1013904223u);
    return ((*s >> 8) & 0xFFFFFF) / double(0xFFFFFF);
}

/* initialization kernel */
__global__ void k_init(double* orb, double* riv, double* myc, int P, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int p = blockIdx.y;
    if(i < N && p < P){
        int idx = p * N + i;
        uint32_t seed = (uint32_t)(clock64() + idx);
        orb[idx] = 0.9 + 0.05 * lcg(&seed);
        riv[idx] = 0.995 + 0.005 * lcg(&seed);
        myc[idx] = 0.9995 + 0.0005 * lcg(&seed);
    }
}

/* update kernel - simulates pocket evolution */
__global__ void k_update(double* O, double* R, double* M, int P, int N, double jitter, int tick){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int p = blockIdx.y;
    if(i < N && p < P){
        int idx = p * N + i;
        uint32_t seed = (uint32_t)(clock64() + idx + tick);
        double j = (lcg(&seed) - 0.5) * jitter;
        
        double o = O[idx] + 0.0001 + j*0.02;
        double r = R[idx] - 0.0002 + j*0.1;
        double m = M[idx] - 0.00001 + j*0.05;
        
        // clamp
        o = fmin(fmax(o, 0.90), 0.95);
        r = fmin(fmax(r, 0.995), 1.000);
        m = fmin(fmax(m, 0.9995), 1.0000);
        
        O[idx] = o;
        R[idx] = r;
        M[idx] = m;
    }
}

// reduction: mean over pockets for each node
__global__ void k_mean_over_pockets(const double* __restrict__ X, double* __restrict__ OUT, int P, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return;
    double acc = 0.0;
    for(int p = 0; p < P; ++p) acc += X[p * N + i];
    OUT[i] = acc / (double)P;
}

extern "C" int pocket_spawn_and_run(const pocket_cfg_t* cfg){
  if(!cfg || cfg->pockets<=0 || cfg->nodes<=0 || cfg->steps<=0) return -1;
  G_P = cfg->pockets; G_N = cfg->nodes;

  size_t PN = (size_t)G_P * (size_t)G_N;
  CUDA_OK(hipMalloc(&d_orb, PN*sizeof(double)));
  CUDA_OK(hipMalloc(&d_riv, PN*sizeof(double)));
  CUDA_OK(hipMalloc(&d_myc, PN*sizeof(double)));
  CUDA_OK(hipMalloc(&d_orb_mean, G_N*sizeof(double)));
  CUDA_OK(hipMalloc(&d_riv_mean, G_N*sizeof(double)));
  CUDA_OK(hipMalloc(&d_myc_mean, G_N*sizeof(double)));

  dim3 block(128,1);              // nodes x pockets split
  dim3 grid((G_N+block.x-1)/block.x, G_P); // one warp-row per pocket

  k_init<<<grid, block>>>(d_orb, d_riv, d_myc, G_P, G_N);
  CUDA_OK(hipGetLastError());

  // stream per pocket (optional; kernel already 2D). Example multi-stream loop:
  // Here we keep single kernel per tick for simplicity and good occupancy.
  for(int t=0;t<cfg->steps;++t){
    k_update<<<grid, block>>>(d_orb, d_riv, d_myc, G_P, G_N, cfg->jitter, t);
    CUDA_OK(hipGetLastError());
  }
  CUDA_OK(hipDeviceSynchronize());

  // means per node
  dim3 block1(256);
  dim3 grid1((G_N+block1.x-1)/block1.x);
  k_mean_over_pockets<<<grid1, block1>>>(d_orb, d_orb_mean, G_P, G_N);
  k_mean_over_pockets<<<grid1, block1>>>(d_riv, d_riv_mean, G_P, G_N);
  k_mean_over_pockets<<<grid1, block1>>>(d_myc, d_myc_mean, G_P, G_N);
  CUDA_OK(hipGetLastError());
  CUDA_OK(hipDeviceSynchronize());
  return 0;
}

extern "C" int pocket_merge_to_host(double* orbital, double* river, double* mycelial){
  if(!d_orb_mean||!d_riv_mean||!d_myc_mean) return -1;
  CUDA_OK(hipMemcpy(orbital,  d_orb_mean, G_N*sizeof(double), hipMemcpyDeviceToHost));
  CUDA_OK(hipMemcpy(river,    d_riv_mean, G_N*sizeof(double), hipMemcpyDeviceToHost));
  CUDA_OK(hipMemcpy(mycelial, d_myc_mean, G_N*sizeof(double), hipMemcpyDeviceToHost));
  return 0;
}

extern "C" int pocket_release(){
  if(d_orb)       hipFree(d_orb), d_orb=nullptr;
  if(d_riv)       hipFree(d_riv), d_riv=nullptr;
  if(d_myc)       hipFree(d_myc), d_myc=nullptr;
  if(d_orb_mean)  hipFree(d_orb_mean), d_orb_mean=nullptr;
  if(d_riv_mean)  hipFree(d_riv_mean), d_riv_mean=nullptr;
  if(d_myc_mean)  hipFree(d_myc_mean), d_myc_mean=nullptr;
  G_P=G_N=0;
  return 0;
}
